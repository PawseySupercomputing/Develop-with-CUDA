#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

// prints error if detected and exits 
void inline check(hipError_t err, const char* filename, int line)
{
	if (err != hipSuccess) 
	{ 
		printf("%s-l%i: %s\n", filename, line, hipGetErrorString(err)); 
		exit(EXIT_FAILURE);
	}
}

// detects cublas non-sucess status and exits
void inline checkStatus(hipblasStatus_t status, const char* filename, int line)
{
   if (status != HIPBLAS_STATUS_SUCCESS)
  {
        printf("%s-l%i: cublas status %i\n", filename, line, status);
        exit(EXIT_FAILURE);
  }
}

// prints start and end of float array
void printArrayTerse(float* array, int length, int num)
{
	if (length<2*num) { num = length/2; }
	for (int i=0; i<num; i++)
	{
		printf("%.0f ",array[i]);
	}
	printf("... ");
    for (int i=length-num; i<length; i++)
    {
        printf("%.0f ",array[i]);
    }
    printf("\n");

}

// add two vectors
int main(int argc, char** argv)
{
	// variable declarations
	hipError_t err;                 // variable for error codes
    hipblasStatus_t status;           // variable for cublas status
    hipblasHandle_t handle;           // variable for cublas handle
    int device;                      // current device id
    struct hipDeviceProp_t prop;      // current device properties
	float* hostArrayA;                 // pointer for array A in host memory
    float* hostArrayB;                 // pointer for array B in host memory
	float* deviceArrayA;               // pointer for array A in device memory
    float* deviceArrayB;               // pointer for array B in device memory
	int length = 262144;             // length of array
    int size = length*sizeof(float);   // size of array in bytes

    // get device properties
    err = hipGetDevice(&device);
    check(err, __FILE__, __LINE__);
    err = hipGetDeviceProperties(&prop, device);
    check(err, __FILE__, __LINE__);
    printf("\nDevice properties: using %s\n\n",prop.name);

	// allocate host memory
	err = hipHostAlloc((void**)&hostArrayA,size,hipHostMallocDefault);
	check(err, __FILE__, __LINE__);
    err = hipHostAlloc((void**)&hostArrayB,size,hipHostMallocDefault);
    check(err, __FILE__, __LINE__);

	// allocate device memory
	err = hipMalloc((void**)&deviceArrayA,size);
    check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&deviceArrayB,size);
    check(err, __FILE__, __LINE__);

	// initialise host memory
	for(int i=0; i<length; i++)
	{
		hostArrayA[i] = i;
        hostArrayB[i] = 1;
	}

    // print host memory values for all arrays
	printf("Array A: ");
	printArrayTerse(hostArrayA,length,8);
    printf("Array B: ");
    printArrayTerse(hostArrayB,length,8);

	// prepare cuBLAS context
    status = hipblasCreate(&handle);
    checkStatus(status,__FILE__,__LINE__);

	// copy host to device for arrays A and B
	status = hipblasSetVector(length,sizeof(float),hostArrayA, 1, deviceArrayA, 1);
    checkStatus(status, __FILE__, __LINE__);
    // HINT: use cublasSetVector to copy array B to the device
    checkStatus(status, __FILE__, __LINE__);
    printf("\nCopied array A and B to device\n\n");

    // perform B = 1*A + B using cublas
	const float c = 1.0f;
    // HINT: use cublasSaxpy to at array A and B together
    checkStatus(status, __FILE__, __LINE__);
    printf("Performed B = A + B using cublas\n\n");

	// copy device to host for array B
	// HINT: use cublasGetVector copy array B back to the host
    checkStatus(status, __FILE__, __LINE__);
    printf("Copied array B from device\n\n");

    // destroy cuBLAS context
    status = hipblasDestroy(handle);
    checkStatus(status,__FILE__,__LINE__);

	// print host memory values for array C
    printf("Array B: ");
    printArrayTerse(hostArrayB,length,8);

	// free device memory
    err = hipFree(deviceArrayA);
    check(err, __FILE__, __LINE__);
    err = hipFree(deviceArrayB);
    check(err, __FILE__, __LINE__);

	// free host memory
	err = hipHostFree(hostArrayA);
    check(err, __FILE__, __LINE__);
    err = hipHostFree(hostArrayB);
    check(err, __FILE__, __LINE__);
    printf("\nFreed device and host memory\n\n");

	// exit
	return EXIT_SUCCESS;
}
